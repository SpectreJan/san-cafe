#include "hip/hip_runtime.h"
#include <filterbank_execute.h>
#include <pfb.h>
#include <stream_to_streams.h>
#include <iostream>

namespace cuda
{
void pfb_execute(float2 *in_stream, float2 *in_streams, float2 *fft_in,
                 float2 *fft_out, pfb_cuda_config *cuda_config,
                 int channel_size, int history,
                 size_t shared_mem_size)
{
  dim3 shuffle_blockconfig(cuda_config->shuffle_blockdim_x,
                           cuda_config->shuffle_blockdim_y, 1);
  dim3 fb_blockconfig(cuda_config->fb_blockdim_x, cuda_config->fb_blockdim_y,
                      1);
  stream_to_streams<<<cuda_config->shuffle_griddim, shuffle_blockconfig>>>(
      in_stream, in_streams, channel_size, history);
  hipDeviceSynchronize();
  hipError_t err = hipGetLastError();

  if (err != hipSuccess) {
    std::cout << "Shuffle kernel failed with message : "
              << hipGetErrorString(err) << "\n";
  }

  filterbank_execute<<<cuda_config->fb_griddim, fb_blockconfig,
                       shared_mem_size>>>(in_streams, fft_in);
  hipDeviceSynchronize();

  err = hipGetLastError();

  if (err != hipSuccess) {
    std::cout << "Filterbank kernel failed with message : "
              << hipGetErrorString(err) << "\n";
  }
}

} /* end namespace cuda */
